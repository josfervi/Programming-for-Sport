
#include <hip/hip_runtime.h>
# include <stdio.h>

// The KERNEL itself: THE GPU CODE.
// 
// looks like a serial program that will run on one thread
// the CPU is responsible for launching the program on many parallel threads
__global__ void square(float *d_out, float *d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx]; // do you need this intermediate var?
    d_out[idx] = f * f;
}


// another Kernel
__global__ void cube(float * d_out, float * d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f * f;
}

// What lies below is THE CPU CODE.
int main(int argc, char **argv) {
    const int ARRAY_SIZE = 64;
    const int ARRAY_NUM_BYTES = ARRAY_SIZE * sizeof(float);
    
    // generate the input array on the host/CPU
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];
    
    // declare GPU memory pointers
    float * d_in;
    float * d_out;
    
    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_NUM_BYTES);
    hipMalloc((void **) &d_out, ARRAY_NUM_BYTES);
    
    // transfer the data from the CPU to the GPU
    hipMemcpy(d_in, h_in, ARRAY_NUM_BYTES, hipMemcpyHostToDevice);
    
    // launch the kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);
    // cube<<<1, ARRAY_SIZE>>>(d_out, d_in);
    
    // copy back the result data from the GPU to the CPU
    hipMemcpy(h_out, d_out, ARRAY_NUM_BYTES, hipMemcpyDeviceToHost);
    
    // print out the resulting array
    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf( ((i % 4) == 3) ? "\n" : "\t" );
    }
    
    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
    
    return 0;
}