
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
    printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}


int main(int argc,char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");

    return 0;
}

// Quiz:
//   how many different outputs
//   can different runs of this
//   program produce?
//
// Answer:
//   16! ~ 21 trillion
//   for the reasons given:
//     CUDA does makes few guarantees
//     about when and where thread
//     blocks will run.