// not verified


#include <hip/hip_runtime.h>
#include <stdio.h>

#define ARRAY_SIZE 128

__global__ void kernel()
{
	int idx = threadIdx.x;
	
	// declare a shared variable
	// shared by all the threads
	// in this thread block
	// Q: does each thread have to do this?
	__shared__ int array[ARRAY_SIZE];
	
	array[idx] = idx; // write op
	
	// all writes should complete before we allow any reads
	__syncthreads();
	
	if (idx < ARRAY_SIZE-1)
	{
	    int temp = array[idx + 1]; // read op
	    
	    // all reads should complete before we allow any writes
	    __syncthreads();
	    
	    array[idx] = temp;
	    
	    // to be safe
	    // all writes should complete before we move on
	    __syncthreads();
	}
}

int main(int argc, char **argv) {
    
    // launch the kernel
    kernel<<<1, ARRAY_SIZE>>>();
    
    return 0;
}